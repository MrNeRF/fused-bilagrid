#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

__global__ void tv_loss_forward_kernel(
    const float* __restrict__ bilagrid,  // [N,12,L,H,W]
    float* __restrict__ tv_loss,
    int N, int L, int H, int W
) {
    // Use 1D grid for better load balancing
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int total = N * L * H * W;
    
    float local_sum = 0.0f;
    
    // Grid-stride loop
    for (int idx = tid; idx < total; idx += stride) {
        // Decode position
        int tmp = idx;
        int wi = tmp % W; tmp /= W;
        int hi = tmp % H; tmp /= H;
        int li = tmp % L; tmp /= L;
        int ni = tmp;
        
        // Process all 12 channels
        #pragma unroll 12
        for (int ci = 0; ci < 12; ci++) {
            int base = (ni*12+ci)*L*H*W;
            int cell_idx = base + (li*H+hi)*W+wi;
            
            float val = bilagrid[cell_idx];
            
            // X-direction
            if (wi > 0) {
                float val0 = bilagrid[cell_idx - 1];
                float diff = val - val0;
                local_sum += diff * diff / (L*H*(W-1));
            }
            
            // Y-direction
            if (hi > 0) {
                float val0 = bilagrid[cell_idx - W];
                float diff = val - val0;
                local_sum += diff * diff / (L*(H-1)*W);
            }
            
            // Z-direction
            if (li > 0) {
                float val0 = bilagrid[cell_idx - W*H];
                float diff = val - val0;
                local_sum += diff * diff / ((L-1)*H*W);
            }
        }
    }
    
    local_sum /= (12*N);
    
    // Warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    local_sum = cg::reduce(warp, local_sum, cg::plus<float>());
    
    // First thread in each warp adds to global result
    if (warp.thread_rank() == 0) {
        atomicAdd(tv_loss, local_sum);
    }
}

void tv_loss_forward(
    const float* bilagrid,
    float* tv_loss,
    int N, int L, int H, int W
) {
    int threads = 256;
    int total = N * L * H * W;
    int blocks = min((total + threads - 1) / threads, 2048);
    
    tv_loss_forward_kernel<<<blocks, threads>>>(
        bilagrid, tv_loss,
        N, L, H, W
    );
}