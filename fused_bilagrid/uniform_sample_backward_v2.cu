#include "hip/hip_runtime.h"
#include "config.h"


__global__ void bilagrid_uniform_sample_backward_v2_kernel(
    const float* __restrict__ bilagrid,  // [N,12,L,H,W]
    const float* __restrict__ rgb,  // [N,m,h,w,3]
    const float* __restrict__ v_output,  // [N,m,h,w,3]
    float* __restrict__ v_bilagrid,  // [N,12,L,H,W]
    float* __restrict__ v_rgb,  // [N,m,h,w,3]
    int N, int L, int H, int W,
    int m, int h, int w
) {
    #if 0
    int wi = blockIdx.x * blockDim.x + threadIdx.x;
    int hi = blockIdx.y * blockDim.y + threadIdx.y;
    #else
    // reduces number of threads writing to the same address at a time in atomicAdd
    int wi = threadIdx.x * ((w+blockDim.x-1) / blockDim.x) + blockIdx.x;
    int hi = threadIdx.y * ((h+blockDim.y-1) / blockDim.y) + blockIdx.y;
    #endif

    int idx = blockIdx.z * blockDim.z + threadIdx.z;
    bool inside = (wi < w && hi < h && idx < (N*m));
    if (!inside) return;
    int mi = idx % m;
    int ni = idx / m;

    // grid coords
    int g_off = (((ni*m + mi)*h + hi)*w + wi) * 3;
    float sr = rgb[g_off+0], sg = rgb[g_off+1], sb = rgb[g_off+2];
    float x = (float)wi / (float)(w-1) * (float)(W-1);
    float y = (float)hi / (float)(h-1) * (float)(H-1);
    float z = (kC2G_r * sr + kC2G_g * sg + kC2G_b * sb) * (L-1);

    // floor + ceil, clamped
    int x0 = floorf(x), y0 = floorf(y), z0 = floorf(z);
    int x1 = x0 + 1, y1 = y0 + 1, z1 = z0 + 1;
    z0 = min(max(z0,0), L-1); z1 = min(max(z1,0), L-1);

    // fractional parts
    float fx = x - x0, fy = y - y0, fz = z - z0;

    // read rgb coeffs and upstream gradient
    float dr = v_output[g_off+0];
    float dg = v_output[g_off+1];
    float db = v_output[g_off+2];
    float vr = 0.0, vg = 0.0, vb = 0.0;

    // spatial derivatives for coords

    float gz_grad = 0.f;

    #pragma unroll
    for (int corner = 0; corner < 8; ++corner) {
        int xi = (corner & 1) ? x1 : x0;
        int yi = (corner & 2) ? y1 : y0;
        int zi = (corner & 4) ? z1 : z0;

        float dfdz = ((corner & 1) ? fx : (1-fx)) *
            ((corner & 2) ? fy : (1-fy)) * ((corner & 4) ? 1 : -1);
        float f = dfdz * ((corner & 4) ? fz : (fz-1));

        float trilerp = 0.f;
        #pragma unroll
        for (int ci = 0; ci < 12; ++ci) {
            int bidx = (((ni*12 + ci)*L + zi)*H + yi)*W + xi;
            int si = ci % 4, di = ci / 4;

            float r_coeff = (si==0 ? sr : si==1 ? sg : si==2 ? sb : 1.f);
            float gout = (di==0 ? dr : di==1 ? dg : db);

            float v = bilagrid[bidx];

            if (si < 3)
                (si == 0 ? vr : si == 1 ? vg : vb) += v * f * gout;

            float grad_weight = r_coeff * gout;
            trilerp += v * grad_weight;
            atomicAdd(v_bilagrid+bidx, f * grad_weight);
        }
        gz_grad += dfdz * (L-1) * trilerp;
    }

    // save gradient, with discontinuity masking
    gz_grad *= (float)(z0 != z && z1 != z);
    v_rgb[g_off+0] = vr + kC2G_r * gz_grad;
    v_rgb[g_off+1] = vg + kC2G_g * gz_grad;
    v_rgb[g_off+2] = vb + kC2G_b * gz_grad;
}


void bilagrid_uniform_sample_backward_v2(
    const float* bilagrid,
    const float* rgb,
    const float* v_output,
    float* v_bilagrid,
    float* v_rgb,
    int N, int L, int H, int W,
    int m, int h, int w
) {
    dim3 block = { 16, 16, 1 };
    dim3 bounds = {
        (w +block.x-1)/block.x,
        (h +block.y-1)/block.y,
        (N*m +block.z-1)/block.z
    };
    bilagrid_uniform_sample_backward_v2_kernel<<<bounds, block>>>(
        bilagrid, rgb, v_output,
        v_bilagrid, v_rgb,
        N, L, H, W, m, h, w
    );
}
