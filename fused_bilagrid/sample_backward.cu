#include "hip/hip_runtime.h"
#include "sample_backward_kernel.cu"

#define COMPUTE_COORDS_GRAD
#include "sample_backward_kernel.cu"

void bilagrid_sample_backward(
    const float* bilagrid,
    const float* coords,
    const float* rgb,
    const float* v_output,
    float* v_bilagrid,
    float* v_coords,
    float* v_rgb,
    int N, int L, int H, int W,
    int m, int h, int w
) {
    // Use 1D grid for better load balancing
    int threads = 256;
    int total = N * m * h * w;
    int blocks = min((total + threads - 1) / threads, 65535);
    
    if (v_coords == nullptr) {
        bilagrid_sample_backward_kernel<<<blocks, threads>>>(
            bilagrid, coords, rgb, v_output,
            v_bilagrid, v_rgb,
            N, L, H, W, m, h, w
        );
    }
    else {
        bilagrid_sample_backward_kernel_cg<<<blocks, threads>>>(
            bilagrid, coords, rgb, v_output,
            v_bilagrid, v_coords, v_rgb,
            N, L, H, W, m, h, w
        );
    }
}
